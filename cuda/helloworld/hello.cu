
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void helloFromGPU() {
    int t = threadIdx.x;
    printf("Hello World from GPU %d!\n", t);
}

int main() {
    printf("Hello World from CPU!\n");

    helloFromGPU <<<1,10>>>();
    //cudaDeviceReset();

    return 0;
}